#include "hip/hip_runtime.h"
#include "bm.cuh"
#include "common.cuh"
#include <math.h>
#include <stdlib.h>
#include <memory.h>

void bmBuildCpu(int** badCharPtr, int** goodSuffixPtr)
{
	*badCharPtr = (int*)malloc(ALPHABET_SIZE * sizeof(int));
	*goodSuffixPtr = (int*)malloc(patternLen * sizeof(int));
	int* maxSuffix = (int*)malloc(patternLen * sizeof(int));

	int* badChar = *badCharPtr;
	int* goodSuffix = *goodSuffixPtr;
	memset(badChar, 0xff, sizeof(int) * ALPHABET_SIZE);
	memset(goodSuffix, 0xff, sizeof(int) * patternLen);

	// ��ǰ����˳�����������Ӧ�ַ�λ�õ�badCharֵ�޸�Ϊ��ǰiֵ
	// ͨ�����ַ�����֤badChar�д洢����ĳ���ַ������ֵ�λ��
	// ƥ���������תʱ���Ա���Ӧ��ת���ȹ������¶�ʧƥ��
	for (int i = 0; i < patternLen; ++i)
	{
		badChar[pattern[i]] = i;
	}

	// ��maxSuffix�е�ÿ��λ��i��¼��ֵ�������Ӵ�[0-i]�ĺ�׺ƥ���
	// �ģʽ����׺�ĳ���
	maxSuffix[patternLen - 1] = patternLen;
	for (int i = 0; i < patternLen - 1; ++i)
	{
		int j = i;
		int k = patternLen - 1;
		int cnt = 0;

		while (j >= 0 && pattern[j]==pattern[k])
		{
			++cnt;
			--j;
			--k;
		}

		maxSuffix[i] = cnt;
	}

	// ͨ��maxSuffix�м�¼��ֵ���Ժ����׵ļ����
	// ģʽ����ÿ����׺��һ����ģʽ���г��ֵ�λ��
	// ��û�г�����Ϊ��ʼ��ֵ-1
	// ��¼���һ�γ���λ�õ�ԭ���뻵�ַ����鹹��������ͬ
	for (int i = 0; i < patternLen - 1; ++i)
	{
		if (maxSuffix[i] > 0)
		{
			goodSuffix[patternLen - maxSuffix[i]] = i;
		}
	}

	free(maxSuffix);
}

void bmCpu()
{
	int* badChar;
	int* goodSuffix;
	bmBuildCpu(&badChar, &goodSuffix);

	int* matchNum;
	int* matchIdx;
	patternMatchCpuMalloc(&matchNum, &matchIdx);

	int i = patternLen - 1;

	while (i < textLen)
	{
		int j = patternLen - 1;
		int k = i;
		int len = 0;

		// ƥ��ʱ�Ӻ���ǰƥ��
		while (j>=0 && text[k] == pattern[j])
		{
			--k;
			--j;
		}

		if (j < 0)
		{
			++k;
			matchIdx[*matchNum] = k;
			++(*matchNum);
			++i;
		}
		else
		{
			// ��ƥ��ʧ�ܣ���ʹ�û��ַ���ת������ú�׺��ת����
			// ѡȡ��ת���Ƚϴ���ִ����ת
			// ��������ת���Ⱦ�С��1����ת������Ϊ1

			// ���ַ����Խ���ǰλ���뵱ǰ�ı�λ�õ��ַ���ģʽ�������һ�γ��ֵ�λ��
			// �����Ϊ��ת���ȣ�������ת֮���ģʽ���ַ��뵱ǰ�ı�λ�õ��ַ����
			// �ٴ��ɺ���ǰƥ�伴��
			// �����������תλ�ÿ���Ϊ����������������ת��С����Ϊ1
			int largestGoodSuffixPos = -1;
			int offset = __max(1, j - badChar[text[k]]);

			// �ú�׺���Դ����Ѿ�ƥ��ɹ����ַ����������ɺ���ǰƥ��
			// ��ƥ��ɹ����ַ�����ģʽ���ĺ�׺
			// ����׺��֮ǰ��ģʽ����δ���֣������̺�׺����ֱ������Ϊֹ
			// ��ģʽ��β��λ�ü�ȥ��׺��һ�γ���λ����Ϊ��ת����
			// �������Ա�֤֮ǰ�Ѿ�ƥ��ĺ�׺û�б��˷�
			// ��û��һ����׺���ֹ����൱��û�ж���תλ�ý��д���
			for (int l = j + 1; l < patternLen - 1; ++l)
			{
				if (goodSuffix[l] != -1)
				{
					offset = __max(offset, patternLen - 1 - goodSuffix[l]);
					break;
				}
			}

			i += offset;
		}
	}

#ifdef PRINT
	printMatchOutputCpu(matchNum, matchIdx);
#endif

	free(badChar);
	free(goodSuffix);
	patternMatchCpuFree(matchNum, matchIdx);
}

void __global__ bmBadCharKernel(const char* __restrict__ pattern, const int patternLen, int* badChar)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < patternLen)
	{
		// ���м����ַ����ֵ����λ��ʱ��ֹд���ͻ
		// ���ﲻ��Ҫʹ��˽��ԭ�Ӳ�������Ϊֻ����һ��д��
		atomicMax(badChar + pattern[idx], idx);
	}
}

void __global__ bmGoodSuffixKernel(const char* __restrict__ pattern, const int patternLen, int* goodSuffix)
{
	extern __shared__ char sharedPattern[];
	int perThreadPatternLen = ceil(patternLen * 1.0 / blockDim.x);

	for (int i = threadIdx.x * perThreadPatternLen; i < (threadIdx.x + 1) * perThreadPatternLen && i < patternLen; ++i)
	{
		sharedPattern[i] = pattern[i];
	}

	__syncthreads();

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < patternLen - 1)
	{
		int cnt = 0;
		int j = idx;
		int k = patternLen - 1;

		while (j >= 0 && sharedPattern[j] == sharedPattern[k])
		{
			++cnt;
			--j;
			--k;
		}

		if (cnt > 0)
		{
			// ͳ�Ƴ��Ӵ�[0,idx]��ģʽ����׺���ƥ�䳤��֮��
			// �������бȽϲ�ִ��ԭ��д�룬����ȴ�maxSuffixȫ���������
			// ģʽ�����Ƚ�С��ʹ��˽��ԭ�Ӳ�����������
			atomicMax(goodSuffix + patternLen - cnt, idx);
		}
	}
}

void __global__ bmKernel(const char* __restrict__ text, const int textLen, const int threadTextLen, const char* __restrict__ pattern, const int patternLen, const int* __restrict__ badChar, const int* __restrict__ goodSuffix, int* matchNum, int* matchIdx)
{
	// ����ÿ���̶߳�������һ���ı��������ڴ��޷�����ÿ��block���账���ı����ʲ����ı����ؽ������ڴ�
	// ����block�����ı���������ƥ����Ҳ�ϴ󣬹��޷�����˽�л�ԭ�Ӳ���
	__shared__ int sharedBadChar[ALPHABET_SIZE];
	extern __shared__ char sharedMemory[];
	int* sharedGoodSuffix = (int*)(sharedMemory);
	char* sharedPattern = (char*)(sharedGoodSuffix + patternLen);

	int perThreadBadCharLen = ceil(ALPHABET_SIZE * 1.0 / blockDim.x);
	int perThreadPatternLen = ceil(patternLen * 1.0 / blockDim.x);

	for (int i = threadIdx.x * perThreadBadCharLen; i < (threadIdx.x + 1) * perThreadBadCharLen && i < ALPHABET_SIZE; ++i)
	{
		sharedBadChar[i] = badChar[i];
	}

	for (int i = threadIdx.x * perThreadPatternLen; i < (threadIdx.x + 1) * perThreadPatternLen && i < patternLen; ++i)
	{
		sharedPattern[i] = pattern[i];
		sharedGoodSuffix[i] = goodSuffix[i];
	}

	__syncthreads();

	// Ϊȷ���ҵ�����ƥ�䣬ÿ���̶߳�����ģʽ������-1���ַ�
	int extendedThreadTextLen = threadTextLen + patternLen - 1;
	int textIdx = (blockIdx.x * blockDim.x + threadIdx.x) * threadTextLen;
	int i = patternLen - 1;

	while (i < extendedThreadTextLen && textIdx + i < textLen)
	{
		int j = patternLen - 1;
		int k = i;
		int len = 0;
		
		while (j >= 0 && text[textIdx + k] == sharedPattern[j])
		{
			--k;
			--j;
		}

		if (j < 0)
		{
			++k;
			int idx = atomicAdd(matchNum, 1);
			matchIdx[idx] = textIdx + k;
			++i;
		}
		else
		{
			int largestGoodSuffixPos = -1;
			int offset = __max(1, j - sharedBadChar[text[textIdx + k]]);

			for (int l = j + 1; l < patternLen - 1; ++l)
			{
				if (sharedGoodSuffix[l] != -1)
				{
					offset = __max(offset, patternLen - 1 - sharedGoodSuffix[l]);
					break;
				}
			}

			i += offset;
		}
	}
}

void bmBuildGpu(int** badCharDevPtr, int** goodSuffixDevPtr)
{
	dim3 blockSize = blockLen;
	dim3 gridSize = ceil(patternLen * 1.0 / blockSize.x);

	hipMalloc(badCharDevPtr, sizeof(int) * ALPHABET_SIZE);
	hipMemset(*badCharDevPtr, 0xff, sizeof(int) * ALPHABET_SIZE);
	bmBadCharKernel <<< gridSize, blockSize >>> (patternDev, patternLen, *badCharDevPtr);
	
	hipMalloc(goodSuffixDevPtr, sizeof(int) * patternLen);
	hipMemset(*goodSuffixDevPtr, 0xff, sizeof(int) * patternLen);
	bmGoodSuffixKernel <<< gridSize, blockSize, patternLen >>> (patternDev, patternLen, *goodSuffixDevPtr);	
}

void bmGpu()
{
	dim3 blockSize = blockLen;
	dim3 gridSize = ceil(textLen * 1.0 / (blockSize.x * threadTextLen));

	int* matchNumDev = nullptr;
	int* matchIdxDev = nullptr;
	patternMatchGpuMalloc(&matchNumDev, &matchIdxDev);

	int* badCharDev = nullptr;
	int* goodSuffixDev = nullptr;
	bmBuildGpu(&badCharDev, &goodSuffixDev);

	bmKernel <<< gridSize, blockSize, patternLen * (sizeof(char) + sizeof(int)) >>> (textDev, textLen, threadTextLen, patternDev, patternLen, badCharDev, goodSuffixDev, matchNumDev, matchIdxDev);

#ifdef PRINT
	printMatchOutputGpu(matchNumDev, matchIdxDev);
#endif

	hipFree(badCharDev);
	hipFree(goodSuffixDev);
	patternMatchGpuFree(matchNumDev, matchIdxDev);
}
