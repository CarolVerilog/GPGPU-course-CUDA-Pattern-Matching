#include "hip/hip_runtime.h"
#include "ssef.cuh"
#include "common.cuh"
#include <stdlib.h>
#include <memory.h>
#include <math.h>
#include <immintrin.h>
#include <time.h>
#include <stdio.h>

#define ASCII_LEN 8
#define MAX_FILTER 65536

typedef struct SsefNode
{
	SsefNode* next = NULL;
	int idx = 1;
} SsefNode;

SsefNode filter[MAX_FILTER];
int filterLen;
int optimalOffset;
int validPatternLen;

void ssefGetOptimalOffsetCpu()
{
	// ͳ����ӽ�0��1��ռ50%��bitλ
	// ʹ�ô�λ���ɹ�ϣ����

	int bit[ASCII_LEN] = { 0 };

	for (int i = 0; i < patternLen; ++i)
	{
		for (int j = 0; j < ASCII_LEN; ++j)
		{
			bit[j] += (pattern[i] >> j) & 0x1;
		}
	}
	
	int minDist = INT_MAX;
	int optimalBit = 0;

	for (int i = 0; i < ASCII_LEN; ++i)
	{
		bit[i] = fabs(bit[i] - patternLen / 2);
		if (bit[i] < minDist)
		{
			minDist = bit[i];
			optimalBit = i;
		}
	}

	optimalOffset = ASCII_LEN - 1 - optimalBit;
}

void ssefBuildCpu()
{
	// ����һ��SIMD�Ĵ���������16���ַ�
	// �ʹ�ϣ���볤��Ϊ16
	ssefGetOptimalOffsetCpu();
	filterLen = 16;
	validPatternLen = patternLen - 15;

	for (int i = 0; i < validPatternLen; ++i)
	{
		// ��ÿ���ַ�����ѱ���λ���Ƶ�����λ��
		// ��16���ַ��ķ���λ��ɹ�ϣ����
		auto simdPattern = _mm_loadu_si128((const __m128i*)&pattern[i]);
		auto tmp128 = _mm_slli_epi64(simdPattern, optimalOffset);
		auto f = _mm_movemask_epi8(tmp128);

		// ����λ�ü�¼����ʽ��ϣ���Ӧ��λ����
		SsefNode* node = &filter[f];
		while (node->next != NULL)
		{
			node = node->next;
		}

		node->next = (SsefNode*)malloc(sizeof(SsefNode));
		node->next->next = NULL;
		node->next->idx = i;
	}
}

void ssefFree()
{
	for (int i = 0; i < MAX_FILTER; ++i)
	{
		while (filter[i].next != NULL)
		{
			SsefNode* node = filter[i].next;
			filter[i].next = filter[i].next->next;
			free(node);
		}
	}
}

void ssefCpu()
{	
	// SSEF�㷨˼·���£�
	// ����SIMD�Ĵ����������16���ַ����ʽ��ı�����16���ַ��ֿ飬��0��ʼ���
	// ����ģʽ������Ϊm����m>=32,ģʽ��һ������ռ��ĳһ�飬���ռ��n=floor(m/16)��
	// ����֤��ģʽ����ռ���Ŀ���һ���б��Ϊn-1�ı����Ŀ�
	// ����ÿ�ζ������Ϊn-1�ı����Ŀ飬������ѱ���λ�����ϣֵ��ѯ�Ѿ�����õĹ�ϣ��
	// ����ϣ���д洢��Ԫ�أ������ģʽ���е�ĳ��λ�õĹ�ϣֵ���ֵ��ͬ
	// �������ٽ������ַ��Ƚϼ���

	ssefBuildCpu();
	int searchWindowLen = (floor(patternLen * 1.0 / filterLen) - 1) * filterLen;
	
	int* matchNum;
	int* matchIdx;
	patternMatchCpuMalloc(&matchNum, &matchIdx);
	
	for (int i = searchWindowLen; i < textLen; i += searchWindowLen)
	{
		__m128i simdText = _mm_loadu_si128((const __m128i*) & text[i]);
		__m128i tmp128 = _mm_slli_epi64(simdText, optimalOffset);
		int f = _mm_movemask_epi8(tmp128);

		SsefNode* node = filter[f].next;
		while (node != NULL)
		{
			if (node->idx == 0 || node->idx > searchWindowLen)
			{
				node = node->next;
				continue;
			}

			int j = i - node->idx;
			int k = 0;

			for (; k < patternLen; ++j, ++k)
			{
				if (text[j] != pattern[k])
				{
					break;
				}
			}

			if (k == patternLen)
			{
				matchIdx[*matchNum] = i - node->idx;
				++(*matchNum);
			}

			node = node->next;
		}
	}
	
#ifdef PRINT
	printMatchOutputCpu(matchNum, matchIdx);
#endif

	ssefFree();
	patternMatchCpuFree(matchNum, matchIdx);
}

void __global__ ssefGetOptimalOffsetKernel(const char* __restrict__ pattern, const int patternLen, int* bit)
{
	// 
	__shared__ int sharedBit[ASCII_LEN];
	extern __shared__ char sharedPattern[];
	int perThreadPatternLen = ceil(patternLen * 1.0 / blockDim.x);

	if (threadIdx.x < ASCII_LEN)
	{
		sharedBit[threadIdx.x] = 0;
	}

	for (int i = threadIdx.x * perThreadPatternLen; i < (threadIdx.x + 1) * perThreadPatternLen && i < patternLen; ++i)
	{
		sharedPattern[i] = pattern[i];
	}

	__syncthreads();

	int patternIdx = blockIdx.x * blockDim.x + threadIdx.x;

	if (patternIdx < patternLen)
	{
		for (int i = 0; i < ASCII_LEN; ++i)
		{
			atomicAdd(sharedBit + i, (sharedPattern[threadIdx.x] >> i) & 0x1);
		}

		if (threadIdx.x < ASCII_LEN)
		{
			atomicAdd(bit + threadIdx.x, sharedBit[threadIdx.x]);
		}
	}
}

void __device__ ssefAtomicLinkKernel(const int idx, SsefNode* filter, const int f)
{
	for (int i = 0; i < 32; ++i)
	{
		if (threadIdx.x % 32 != i)
		{
			continue;
		}

		while (atomicExch(&filter[f].idx, 0) == 0);

		SsefNode* node = &filter[f];
		while (node->next != NULL)
		{
			node = node->next;
		}

		node->next = (SsefNode*)malloc(sizeof(SsefNode));
		node->next->next = NULL;
		node->next->idx = idx;

		filter[f].idx = 1;
		return;
	}
}

void __global__ ssefBuildKernel(const char* __restrict__ pattern, const int patternLen, const int filterLen, const int optimalOffset, SsefNode* filter)
{
	extern __shared__ char sharedMovedPattern[];
	int offset = ASCII_LEN - 1 - optimalOffset;
	int validPatternLen = patternLen - filterLen + 1;

	int blockPatternIdx = blockIdx.x * blockDim.x;
	int blockPatternLen = blockDim.x + filterLen - 1;
	int perThreadPatternLen = ceil(blockPatternLen * 1.0 / blockDim.x);

	for (int i = threadIdx.x * perThreadPatternLen; i < (threadIdx.x + 1) * perThreadPatternLen && i < blockPatternLen && blockPatternIdx + i < patternLen; ++i)
	{
		sharedMovedPattern[i] = (pattern[blockPatternIdx + i] >> offset) & 0x1;
	}

	__syncthreads();

	int patternIdx = blockPatternIdx + threadIdx.x;
	if (patternIdx < validPatternLen)
	{
		int f = 0;
		for (int i = 0; i < filterLen; ++i)
		{
			f |= (sharedMovedPattern[threadIdx.x + i] << i);
		}

		ssefAtomicLinkKernel(patternIdx, filter, f);
	}
}

void __global__ ssefKernel(const char* __restrict__ text, const int textLen, const char* __restrict__ pattern, const int patternLen, const SsefNode* __restrict__ filter, const int filterLen, const int optimalOffset, int* matchNum, int* matchIdx)
{
	// ÿ���̸߳���ĳ���ַ�����λ���㲢д�빲���ڴ���
	extern __shared__ char sharedMemory[];
	char* sharedPattern = sharedMemory;
	char* sharedMovedText = sharedMemory + patternLen;
	int perThreadPatternLen = ceil(patternLen * 1.0 / blockDim.x);

	int offset = ASCII_LEN - 1 - optimalOffset;
	int validPatternLen = patternLen - filterLen + 1;
	int searchWindowLen = (floor(patternLen * 1.0 / filterLen) - 1) * filterLen;

	// ���ǽ��̰߳���СΪ16���飬ÿ���߳���λ���������
	// �ɸ����0���߳����ɹ�ϣֵ����ѯ��ϣ��
	// Ϊ�����Ч�ʲ��ý��������˼��
	// �������ͬ���߳̾���������ͬһwarp��ִ��
	// ����ÿ��warp32���߳�����ֻ��2���߳���ִ��
	int windowsCnt = blockDim.x / filterLen;
	int windowIdx = threadIdx.x % windowsCnt;
	int windowThreadIdx = threadIdx.x / windowsCnt;

	int textIdx = (blockIdx.x*windowsCnt + windowIdx + 1) * searchWindowLen + windowThreadIdx;
	int blockTextIdx = windowIdx * filterLen + windowThreadIdx;
	
	for (int i = threadIdx.x * perThreadPatternLen; i < (threadIdx.x + 1) * perThreadPatternLen && i < patternLen; ++i)
	{
		sharedPattern[i] = pattern[i];
	}

	if (threadIdx.x < windowsCnt * filterLen)
	{
		sharedMovedText[blockTextIdx] = (text[textIdx] >> offset) & 0x1;
	}

	__syncthreads();
	
	if (windowThreadIdx == 0 && textIdx < textLen)
	{
		int f = 0;
		for (int i = 0; i < filterLen; ++i)
		{
			f |= sharedMovedText[blockTextIdx + i] << i;
		}
		
		SsefNode* node = filter[f].next;
		while (node != NULL)
		{
			if (node->idx == 0 || node->idx > searchWindowLen)
			{
				node = node->next;
				continue;
			}

			int j = textIdx - node->idx;
			int k = 0;

			for (; k < patternLen; ++j, ++k)
			{
				if (text[j] != pattern[k])
				{
					break;
				}
			}

			if (k == patternLen)
			{
				int idx = atomicAdd(matchNum, 1);
				matchIdx[idx] = textIdx - node->idx;
			}

			node = node->next;
		}
	}
}

void __global__ ssefFreeKernel(SsefNode* filter)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	while (idx < MAX_FILTER && filter[idx].next != NULL)
	{
		SsefNode* node = filter[idx].next;
		filter[idx].next = filter[idx].next->next;
		free(node);
	}
}

void ssefGetOptimalOffsetGpu()
{
	dim3 blockSize = blockLen;
	dim3 gridSize = ceil(patternLen * 1.0 / blockSize.x);

	int* bit = (int*)malloc(ASCII_LEN * sizeof(int));
	int* bitDev = nullptr;

	hipMalloc(&bitDev, ASCII_LEN * sizeof(int));
	hipMemset(bitDev, 0, ASCII_LEN * sizeof(int));
	ssefGetOptimalOffsetKernel <<< gridSize, blockSize, patternLen >>> (patternDev, patternLen, bitDev);
	hipMemcpy(bit, bitDev, ASCII_LEN * sizeof(int), hipMemcpyDeviceToHost);

	int minDist = INT_MAX;
	int optimalBit = 0;

	for (int i = 0; i < ASCII_LEN; ++i)
	{
		bit[i] = fabs(bit[i] - patternLen / 2);
		if (bit[i] < minDist)
		{
			minDist = bit[i];
			optimalBit = i;
		}
	}

	optimalOffset = ASCII_LEN - 1 - optimalBit;
}

void ssefBuildGpu(SsefNode** filterDevPtr)
{
	ssefGetOptimalOffsetGpu();

	dim3 blockSize = blockLen;
	dim3 gridSize = ceil(patternLen * 1.0 / blockSize.x);

	filterLen = patternLen > 32 ? 16 : patternLen / 2;
	hipMalloc(filterDevPtr, MAX_FILTER * sizeof(SsefNode));
	SsefNode* filterDev = *filterDevPtr;

	hipMemcpy(filterDev, filter, MAX_FILTER * sizeof(SsefNode), hipMemcpyHostToDevice);
	ssefBuildKernel <<< gridSize, blockSize, blockSize.x + filterLen - 1 >>> (patternDev, patternLen, filterLen, optimalOffset, filterDev);
}

void ssefGpu()
{
	SsefNode* filterDev;
	ssefBuildGpu(&filterDev);

	int searchWindowLen = (floor(patternLen * 1.0 / filterLen) - 1) * filterLen;
	int windowsCnt = textLen / searchWindowLen;
	int blockWindowsCnt = blockLen / filterLen;

	dim3 blockSize = blockLen;
	dim3 gridSize = ceil(windowsCnt * 1.0 / blockWindowsCnt);

	int* matchNumDev;
	int* matchIdxDev;
	patternMatchGpuMalloc(&matchNumDev, &matchIdxDev);

	ssefKernel <<< gridSize, blockSize, patternLen + blockWindowsCnt * filterLen >>> (textDev, textLen, patternDev, patternLen, filterDev, filterLen, optimalOffset, matchNumDev, matchIdxDev);

#ifdef PRINT
	printMatchOutputGpu(matchNumDev, matchIdxDev);
#endif
	
	gridSize = ceil(MAX_FILTER * 1.0 / blockLen);
	ssefFreeKernel <<< gridSize, blockSize >>> (filterDev);
	hipFree(filterDev);
	patternMatchGpuFree(matchNumDev, matchIdxDev);
}