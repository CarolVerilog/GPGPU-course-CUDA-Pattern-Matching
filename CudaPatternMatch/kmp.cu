#include "hip/hip_runtime.h"
#include "kmp.cuh"
#include "common.cuh"
#include <stdlib.h>
#include <math.h>

void kmpNextCpu(int** nextPtr)
{
	// next�����¼�뵱ǰλ��֮ǰ���Ӵ��ĺ�׺��ͬ��ģʽ��ǰ׺����һ��λ��
	*nextPtr = (int*)malloc(patternLen * sizeof(int));
	int* next = *nextPtr;
	int i = 0;
	int j = -1;
	next[0] = -1;

	while (i < patternLen - 1)
	{
		if (j == -1 || pattern[i] == pattern[j])
		{
			++i, ++j;
			if (pattern[i] != pattern[j])
			{
				// ����ģʽ�����Ӵ�[i-j,i-1]��ǰ׺[0,j-1]��ͬ
				// ��j��Ϊ��ģʽ����iλ��֮ǰ���Ӵ��ĺ�׺��ȵ�ģʽ��ǰ׺�ĺ�һ���ַ�
				next[i] = j;
			}
			else
			{
				// ���ַ����
				// ���ʱ�Ӵ�[i-j,i-1]���Ӵ�[0,j-1]���
				// ��j����next[i]�Ƿ���Ҫ���
				// ����ת֮������i��jλ���ַ���������ı��ַ���Ȼ��ƥ��
				// �ʽ�next[j]����next[i]�������Ż�
				next[i] = next[j];
			}
		}
		else
		{
			// ģʽ�����ȣ���j������next[j]��������
			j = next[j];
		}
	}
}

void kmpCpu()
{
	int* next = nullptr;
	int i = 0;
	int j = 0;
	kmpNextCpu(&next);

	int* matchNum;
	int* matchIdx;
	patternMatchCpuMalloc(&matchNum, &matchIdx);

	while (i < textLen)
	{
		if (j == -1 || text[i] == pattern[j])
		{
			++i, ++j;
		}
		else
		{
			j = next[j];
		}

		if (j == patternLen)
		{
			i -= patternLen;
			j = -1;
			matchIdx[*matchNum] = i;
			++(*matchNum);
		}
	}

#ifdef PRINT
	printMatchOutputCpu(matchNum, matchIdx);
#endif
	
	free(next);
	patternMatchCpuFree(matchNum, matchIdx);
}

void __global__ kmpKernel(const char* __restrict__ text, const int textLen, const int threadTextLen, const char* __restrict__ pattern, const int patternLen, const int* __restrict__ next, int* matchNum, int* matchIdx)
{
	// ����ÿ���̶߳�������һ���ı��������ڴ��޷�����ÿ��block���账���ı����ʲ����ı����ؽ������ڴ�
	// ����block�����ı���������ƥ����Ҳ�ϴ󣬹��޷�����˽�л�ԭ�Ӳ���
	extern __shared__ char sharedMemory[];
	int* sharedNext = (int*)sharedMemory;
	char* sharedPattern = (char*)(sharedNext + patternLen);
	int perThreadPatternLen = ceil(patternLen * 1.0 / blockDim.x);

	for (int i = threadIdx.x * perThreadPatternLen; i < (threadIdx.x + 1) * perThreadPatternLen && i < patternLen; ++i)
	{
		sharedPattern[i] = pattern[i];
		sharedNext[i] = next[i];
	}

	__syncthreads();

	// Ϊȷ���ҵ�����ƥ�䣬ÿ���̶߳�����ģʽ������-1���ַ�
	int extendedThreadTextLen = threadTextLen + patternLen - 1;
	int textIdx = (blockIdx.x * blockDim.x + threadIdx.x) * threadTextLen;

	int i = 0;
	int j = 0;
	
	while (i < extendedThreadTextLen && textIdx + i < textLen)
	{
		if (j == -1 || text[textIdx + i] == sharedPattern[j])
		{
			++i, ++j;
		}
		else
		{
			j = sharedNext[j];
		}

		if (j == patternLen)
		{
			i -= patternLen;
			j = -1;

			int idx = atomicAdd(matchNum, 1);
			matchIdx[idx] = textIdx + i;
		}
	}
}

void kmpGpu()
{
	dim3 blockSize = blockLen;
	dim3 gridSize = ceil(textLen * 1.0 / (blockSize.x * threadTextLen));

	int* next = nullptr;
	int* nextDev = nullptr;

	// ����next��������ɵ�Ԫ�������������ɵ�Ԫ�أ��ʲ�ʹ�ò��л�����
	kmpNextCpu(&next);
	hipMalloc(&nextDev, sizeof(int) * patternLen);
	hipMemcpy(nextDev, next, sizeof(int) * patternLen, hipMemcpyHostToDevice);
	free(next);

	int* matchNumDev = nullptr;
	int* matchIdxDev = nullptr;
	patternMatchGpuMalloc(&matchNumDev, &matchIdxDev);

	kmpKernel <<< gridSize, blockSize, patternLen * (sizeof(char) + sizeof(int)) >>> (textDev, textLen, threadTextLen, patternDev, patternLen, nextDev, matchNumDev, matchIdxDev);

#ifdef PRINT
	printMatchOutputGpu(matchNumDev, matchIdxDev);
#endif

	hipFree(nextDev);
	patternMatchGpuFree(matchNumDev, matchIdxDev);
}
