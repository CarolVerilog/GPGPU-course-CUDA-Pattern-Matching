#include "hip/hip_runtime.h"
#include "epsma.cuh"
#include "common.cuh"
#include <stdlib.h>
#include <memory.h>
#include <math.h>
#include <immintrin.h>

void epsmaBuildCpu(char** extendedPtr)
{
	// ��ģʽ��ÿ���ַ�����16������SIMD�Ƚ�
	// ���罫ģʽ��"ab"��չΪ
	// "aaaaaaaaaaaaaaaabbbbbbbbbbbbbbbb"

	*extendedPtr = (char*)malloc(patternLen * 16);
	char* extended = *extendedPtr;

	for (int i = 0; i < patternLen; ++i)
	{
		memset(extended + i * 16, pattern[i], 16);
	}
}

void epsmaCpu()
{
	char* extended;
	epsmaBuildCpu(&extended);
	
	int* matchNum;
	int* matchIdx;
	patternMatchCpuMalloc(&matchNum, &matchIdx);

	for (int i = 0; i < textLen; i += 16 - patternLen + 1)
	{
		auto simdText = _mm_loadu_si128((const __m128i*)(text + i));
		int matchRes = 0xffff;

		for (int j = 0; j < patternLen; ++j)
		{
			// ��ģʽ��ÿ���ַ�����16�β�����SIMD�Ĵ���
			// �뵱ǰɨ�赽��16������SIMD�Ĵ������ı��ַ����бȽ�
			// �ȽϽ����16bit�洢��������jλ����һ�ν����������
			// ѭ������֮�����ȽϽ����Ϊ0��˵��ƥ��ɹ�

			// ����ab��abcdacccddddabac�Ƚ�
			// ��һ�αȽϽ�a*16���ı��Ƚ�
			// ���Ϊ0101000000010001���ý��Ϊ���Ϊ��λ�ȽϽ�����ұ�Ϊ��λ�ȽϽ����
			// �ڶ��αȽϽ�b*16���ı��Ƚ�
			// ���Ϊ0010000000000010������һλ����һ�ν������ã�0001000000000001
			
			// ���Կ���ƥ���ԭ���ǵ�nΪ�ıȽϽ������nλ��������patternLen�αȽϺ�
			// ��λ����Ȼ��1����˵����λ������patternLen���ַ���ģʽ����ƥ��
			// ����ÿ�ֱȽ϶���Ҫ������ַ�������Ƚ�6��λ�õ�ƥ�䣬ģʽ������Ϊ2
			// ����Ҫ7���ַ��������6���ַ��޷�����ƥ��
			// ��Ϊ�˳������SIMD�Ĵ����Ŀռ䣬ģʽ���������Ϊ16

			auto simdPattern = _mm_loadu_si128((const __m128i*)(extended + j * 16));
			auto cmpRes = _mm_cmpeq_epi8(simdText, simdPattern);
			matchRes &= _mm_movemask_epi8(cmpRes) >> j;

			if (!matchRes)
			{
				break;
			}
		}

		if (matchRes)
		{
			for (int j = 0; j < 16 - patternLen + 1; ++j)
			{
				if (((matchRes >> j) & 0x1) && i + j < textLen - patternLen)
				{
					matchIdx[*matchNum] = i + j;
					++(*matchNum);
				}
			}
		}
	}

#ifdef PRINT
	printMatchOutputCpu(matchNum, matchIdx);
#endif

	free(extended);
	patternMatchCpuFree(matchNum, matchIdx);
}

void __global__ epsmaKernel(const char* __restrict__ text, const int textLen, const char* __restrict__ pattern, const int patternLen, int* matchNum, int* matchIdx)
{
	__shared__ int sharedMatchNum;
	__shared__ int sharedWriteIdx;
	extern __shared__ char sharedMemory[];
	int* sharedMatchIdx = (int*)sharedMemory;
	char* sharedPattern = (char*)(sharedMatchIdx + blockDim.x);
	char* sharedText = sharedPattern + patternLen;

	sharedMatchNum = 0;
	int blockTextIdx = blockIdx.x * blockDim.x;
	int blockTextLen = blockDim.x + patternLen - 1;
	int perThreadPatternLen = ceil(patternLen * 1.0 / blockDim.x);
	int perThreadTextLen = ceil(blockTextLen * 1.0 / blockDim.x);

	for (int i = threadIdx.x * perThreadPatternLen; i < (threadIdx.x + 1) * perThreadPatternLen && i < patternLen; ++i)
	{
		sharedPattern[i] = pattern[i];
	}

	for (int i = threadIdx.x * perThreadTextLen; i < (threadIdx.x + 1) * perThreadTextLen && i < blockTextLen && blockTextIdx + i < textLen; ++i)
	{
		sharedText[i] = text[blockTextIdx + i];
	}

	__syncthreads();

	int textIdx = blockTextIdx + threadIdx.x;
	if (textIdx <= textLen - patternLen)
	{
		int i = 0;
		for (; i < patternLen; ++i)
		{
			if (sharedPattern[i] != sharedText[threadIdx.x + i])
			{
				break;
			}
		}

		if (i == patternLen)
		{
			int idx = atomicAdd(&sharedMatchNum, 1);
			sharedMatchIdx[idx] = textIdx;
		}		
	}

	__syncthreads();

	if (threadIdx.x < sharedMatchNum)
	{
		if (threadIdx.x == 0)
		{
			sharedWriteIdx = atomicAdd(matchNum, sharedMatchNum);
		}

		__syncthreads();

		matchIdx[sharedWriteIdx + threadIdx.x] = sharedMatchIdx[threadIdx.x];
	}

}

void epsmaGpu()
{
	dim3 blockSize = blockLen;
	dim3 gridSize = ceil(textLen * 1.0 / blockSize.x);
	int blockTextLen = blockSize.x + patternLen - 1;

	int* matchNumDev = nullptr;
	int* matchIdxDev = nullptr;
	patternMatchGpuMalloc(&matchNumDev, &matchIdxDev);

	epsmaKernel <<< gridSize, blockSize, patternLen + blockTextLen + blockSize.x * sizeof(int) >>>  (textDev, textLen, patternDev, patternLen, matchNumDev, matchIdxDev);

#ifdef PRINT
	printMatchOutputGpu(matchNumDev, matchIdxDev);
#endif

	patternMatchGpuFree(matchNumDev, matchIdxDev);
}