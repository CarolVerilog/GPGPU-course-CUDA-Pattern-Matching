#include "hip/hip_runtime.h"
#include "sunday.cuh"
#include "common.cuh"
#include <stdlib.h>
#include <memory.h>
#include <math.h>

void sundayMoveCpu(int** movePtr)
{
	*movePtr = (int*)malloc(ALPHABET_SIZE * sizeof(int));
	int* move = *movePtr;
	memset(move, 0xff, sizeof(int) * ALPHABET_SIZE);

	// ��ǰ����˳�����������Ӧ�ַ�λ�õ�moveֵ�޸�Ϊ��ǰiֵ
	// ͨ�����ַ�����֤move�д洢����ĳ���ַ������ֵ�λ��
	// ƥ���������תʱ���Ա���Ӧ��ת���ȹ������¶�ʧƥ��
	for (int i = 0; i < patternLen; ++i)
	{
		move[pattern[i]] = i;
	}
}

void sundayCpu()
{
	int* move;
	sundayMoveCpu(&move);

	int* matchNum;
	int* matchIdx;
	patternMatchCpuMalloc(&matchNum,&matchIdx);

	int i = 0;

	while (i <= textLen - patternLen)
	{
		int j = 0;
		int k = i;

		while (j < patternLen && text[k] == pattern[j])
		{
			++k;
			++j;
		}

		if (j == patternLen)
		{
			matchIdx[*matchNum] = i;
			++(*matchNum);

			i += 1;
		}
		else
		{
			// ��ƥ��ʧ�ܣ���˵���Ӵ�[i,i+patternLen-1]��ģʽ��ƥ��ʧ��
			// ��ʱ��鵱ǰ�ı�λ�ü���ģʽ������֮����ַ�����i+patternLenλ�õ��ַ�
			// ��Ϊ����Ŀ��ܳɹ���ƥ����[i+1,i+patternLen]
			// ���i+patternLenλ�õ��ַ���ģʽ�������һ�γ��ֵ�λ�ò�ִ����ת
			// �������������ת��������ֱ�ӽ�i��ת��i+patternLen+1��Ч�ʺܸ�

			int lastPos = move[text[i + patternLen]];
			int offset = patternLen - lastPos;
			i += offset;
		}
	}

#ifdef PRINT
	printMatchOutputCpu(matchNum, matchIdx);
#endif

	free(move);
	patternMatchCpuFree(matchNum, matchIdx);
}

void __global__ sundayMoveKernel(const char* __restrict__ pattern, const int patternLen, int* move)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < patternLen)
	{
		// ���м����ַ����ֵ����λ��ʱ��ֹд���ͻ
		// ���ﲻ��Ҫʹ��˽��ԭ�Ӳ�������Ϊֻ����һ��д��
		atomicMax(move + pattern[idx], idx);
	}
}

void __global__ sundayKernel(const char* __restrict__ text, const int textLen, const int threadTextLen, const char* __restrict__ pattern, const int patternLen, const int* __restrict__ move, int* matchNum, int* matchIdx)
{
	// ����ÿ���̶߳�������һ���ı��������ڴ��޷�����ÿ��block���账���ı����ʲ����ı����ؽ������ڴ�
	// ����block�����ı���������ƥ����Ҳ�ϴ󣬹��޷�����˽�л�ԭ�Ӳ���
	__shared__ int sharedMove[ALPHABET_SIZE];
	extern __shared__ char sharedPattern[];
	int perThreadMoveLen = ceil(ALPHABET_SIZE * 1.0 / blockDim.x);
	int perThreadPatternLen = ceil(patternLen * 1.0 / blockDim.x);
	
	for (int i = threadIdx.x * perThreadMoveLen; i < (threadIdx.x + 1) * perThreadMoveLen && i < ALPHABET_SIZE; ++i)
	{
		sharedMove[i] = move[i];
	}

	for (int i = threadIdx.x * perThreadPatternLen; i < (threadIdx.x + 1) * perThreadPatternLen && i < patternLen; ++i)
	{
		sharedPattern[i] = pattern[i];
	}

	__syncthreads();

	// Ϊȷ���ҵ�����ƥ�䣬ÿ���̶߳�����ģʽ������-1���ַ�
	int extendedThreadTextLen = threadTextLen + patternLen - 1;
	int textIdx = (blockIdx.x * blockDim.x + threadIdx.x) * threadTextLen;
	int i = 0;

	while (i < threadTextLen && textIdx + i < textLen)
	{
		int j = 0;
		int k = i;

		while (j < patternLen && text[textIdx + k] == sharedPattern[j])
		{
			++k;
			++j;
		}

		if (j == patternLen)
		{
			int idx = atomicAdd(matchNum, 1);
			matchIdx[idx] = textIdx + i;

			i += 1;
		}
		else
		{
			int lastPos = sharedMove[text[textIdx + i + patternLen]];
			int offset = patternLen - lastPos;
			i += offset;
		}
	}
}



void sundayMoveGpu(int** movePtr)
{
	dim3 blockSize = blockLen;
	dim3 gridSize = ceil(patternLen * 1.0 / blockSize.x);

	hipMalloc(movePtr, sizeof(int) * ALPHABET_SIZE);
	hipMemset(*movePtr, 0xff, sizeof(int) * ALPHABET_SIZE);
	sundayMoveKernel <<< gridSize, blockSize, patternLen >>> (patternDev, patternLen, *movePtr);
}

void sundayGpu()
{
	dim3 blockSize = blockLen;
	dim3 gridSize = ceil(textLen * 1.0 / (blockSize.x * threadTextLen));

	int* matchNumDev = nullptr;
	int* matchIdxDev = nullptr;
	patternMatchGpuMalloc(&matchNumDev, &matchIdxDev);

	int* moveDev = nullptr;
	sundayMoveGpu(&moveDev);

	sundayKernel <<< gridSize, blockSize, patternLen >>> (textDev, textLen, threadTextLen, patternDev, patternLen, moveDev, matchNumDev, matchIdxDev);

#ifdef PRINT
	printMatchOutputGpu(matchNumDev, matchIdxDev);
#endif

	hipFree(moveDev);
	patternMatchGpuFree(matchNumDev, matchIdxDev);
}
