#include "hip/hip_runtime.h"
#include "bf.cuh"
#include "common.cuh"
#include <math.h>

void bfCpu()
{
	int* matchNum;
	int* matchIdx;
	patternMatchCpuMalloc(&matchNum, &matchIdx);

	for (int i = 0; i < textLen - patternLen; ++i)
	{
		int j = 0;
		int k = i;

		for (; j < patternLen; ++j, ++k)
		{
			if (text[k] != pattern[j])
			{
				break;
			}
		}

		if (j == patternLen)
		{
			matchIdx[*matchNum] = i;
			++(*matchNum);
		}
	}

#ifdef PRINT
	printMatchOutputCpu(matchNum, matchIdx);
#endif

	patternMatchCpuFree(matchNum, matchIdx);
}

void __global__ bfKernel(const char* __restrict__ text, const int textLen, const char* __restrict__ pattern, const int patternLen, int* matchNum, int* matchIdx)
{
	// ˽�л�ԭ�Ӳ�������
	// ���ڹ����ڴ��㹻����ÿ��block��������ı������ʼ��ؽ������ڴ�
	__shared__ int sharedMatchNum;
	__shared__ int sharedWriteIdx;
	extern __shared__ char sharedMemory[];
	int* sharedMatchIdx = (int*)sharedMemory;
	char* sharedPattern = (char*)(sharedMatchIdx + blockDim.x);
	char* sharedText = sharedPattern + patternLen;
	
	// Ϊȷ����鵽����ƥ�䣬ÿ��block��Ҫ������ģʽ������-1���ַ�
	sharedMatchNum = 0;
	int blockTextIdx = blockIdx.x * blockDim.x;
	int blockTextLen = blockDim.x + patternLen - 1; 	
	int perThreadPatternLen = ceil(patternLen * 1.0 / blockDim.x);
	int perThreadTextLen = ceil(blockTextLen * 1.0 / blockDim.x);

	for (int i = threadIdx.x * perThreadPatternLen; i < (threadIdx.x + 1) * perThreadPatternLen && i < patternLen; ++i)
	{
		sharedPattern[i] = pattern[i];
	}
	
	for (int i = threadIdx.x * perThreadTextLen; i < (threadIdx.x + 1) * perThreadTextLen && i < blockTextLen && blockTextIdx + i < textLen; ++i)
	{
		sharedText[i] = text[blockTextIdx + i];
	}

	__syncthreads();
	
	int textIdx = blockTextIdx + threadIdx.x;
	if (textIdx <= textLen - patternLen)
	{
		int i = 0;
		for (; i < patternLen; ++i)
		{
			if (sharedPattern[i] != sharedText[threadIdx.x + i])
			{
				break;
			}
		}

		if (i == patternLen)
		{
			int idx = atomicAdd(&sharedMatchNum, 1);
			sharedMatchIdx[idx] = textIdx;
		}
	}

	__syncthreads();

	if (threadIdx.x < sharedMatchNum)
	{
		// �̺߳�С��block�ڲ�ƥ���������̸߳������տ���
		// 0���̸߳�������ԭ�Ӳ���
		if (threadIdx.x == 0)
		{
			sharedWriteIdx = atomicAdd(matchNum, sharedMatchNum);
		}

		__syncthreads();

		matchIdx[sharedWriteIdx + threadIdx.x] = sharedMatchIdx[threadIdx.x];
	}
}

void bfGpu()
{
	dim3 blockSize = blockLen;
	dim3 gridSize = ceil(textLen * 1.0 / blockSize.x);
	int blockTextLen = blockSize.x + patternLen - 1;

	int* matchNumDev = nullptr;
	int* matchIdxDev = nullptr;
	patternMatchGpuMalloc(&matchNumDev, &matchIdxDev);

	bfKernel <<< gridSize, blockSize, patternLen + blockTextLen + blockSize.x * sizeof(int) >>>  (textDev, textLen, patternDev, patternLen, matchNumDev, matchIdxDev);

#ifdef PRINT
	printMatchOutputGpu(matchNumDev, matchIdxDev);
#endif

	patternMatchGpuFree(matchNumDev, matchIdxDev);
}
